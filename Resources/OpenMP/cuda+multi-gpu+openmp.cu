#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include<omp.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

void fill_matrix(int *A, int fac, int m, int n)
{
  int i, j;

  for (i=0; i<m;i++)
  {
    for (j=0;j<n;j++)
    {
      A[i*n+j] = i+j*fac;
    }
  }
}

void print_matrix(int *A, int m, int n)
{
  int i, j;

  for (i=0; i<m;i++)
  {
    for (j=0;j<n;j++)
    {
      printf("mat[%d, %d] = %d\n", i, j, A[i*n+j]);
    }
  }
}

//CPU version of the calculations
// just the product c_ij = Aij*B_ij
void perform_operation(int *A, int *B, int *C, int m, int n)
{
  int i, j;

  for (i=0; i<m;i++)
  {
    for (j=0;j<n;j++)
    {
      C[i*n+j] = A[i*n+j]*B[i*n+j];
      //printf("C[%d, %d] = %d\n", i, j, C[i*n+j]);
    }
  }
}

//gpu version of the calculations
__global__ void perform_operation_cuda(int *A, int *B, int *C, int m, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;


  if (i<m)
  {
    if (j<n)
    {
      //printf("Hola!!!, %d, %d\n", i, j);
      //printf("A[%d, %d] = %d\n", i, j, A[i*n+j]);
      C[i*n+j] = A[i*n+j]*B[i*n+j];
    }
  }
}


//do the sum of the different C matrix in order to check the results
int do_sum(int *C, int sum, int m, int n)
{
  int i, j;

  for (i=0; i<m;i++)
  {
    for (j=0;j<n;j++)
    {
      sum = sum + C[i*n+j];
    }
  }

  return sum;
}

struct aux_device
{
  int gpuid; //gpu id
  int *A_d; // device array A on the gpu gpuid
};


//main test program
int main (void)
{
  const int N = 3;
  int A[N*N], B[N*N], C[N*N];
  int f, nf=3, sum = 0, sum_ref = 0;
  int *B_d, *C_d;

  dim3 dimBlock(N*N, N*N);
  dim3 dimGrid(1, 1);

  int num_gpus = -1, num_cpus=-1;
  int gpuid = -1;
  aux_device *dev_mem;
  unsigned int cpu_thread_id = -1;

  //initialisation matrices
  fill_matrix(A, 2, N, N);
  fill_matrix(B, 1, N, N);
  fill_matrix(C, 0, N, N);

  //run for checking
  for (f=0; f<nf; f++)
  {
    fill_matrix(B, f+1, N, N);
    perform_operation(A, B, C, N, N);
    sum_ref = do_sum(C, sum_ref, N, N);
  }
  printf("SUM_REF = %d\n", sum_ref);
  //end references

  
  checkCudaErrors(hipGetDeviceCount(&num_gpus));
  num_cpus = omp_get_max_threads();
  printf("there is %d cpus and %d gpus\n", num_cpus, num_gpus);
  dev_mem = (aux_device *)malloc(num_gpus*sizeof(aux_device));
  //Set the threads to each GPUs

#pragma omp parallel shared(num_gpus, dev_mem) private(cpu_thread_id, gpuid)
{
#pragma omp critical
  {
  cpu_thread_id = omp_get_thread_num();
  checkCudaErrors(hipSetDevice(cpu_thread_id % num_gpus));
  checkCudaErrors(hipGetDevice(&gpuid));
  dev_mem[gpuid].gpuid = cpu_thread_id;
  }
}

  printf("%d, %d, %d\n", dev_mem[0].gpuid, dev_mem[1].gpuid, dev_mem[2].gpuid);


  //allocate and copy the array A ones by GPU
#pragma omp parallel shared(A, dev_mem, num_gpus) private(f, cpu_thread_id, gpuid)
{
#pragma omp critical
  {
    cpu_thread_id = omp_get_thread_num();
    for (f = 0;f<num_gpus;f++)
    {
      if (cpu_thread_id == dev_mem[f].gpuid)
      {
        checkCudaErrors(hipMalloc( (void **)&dev_mem[f].A_d, sizeof(int) * N*N)); //I want it here!!
        checkCudaErrors(hipMemcpy( dev_mem[f].A_d, A, sizeof(int) * N*N, hipMemcpyHostToDevice)); //I want it here!!
        checkCudaErrors(hipGetDevice(&gpuid));
        printf("CPU thread %d uses CUDA device %d\n", cpu_thread_id, gpuid);
        break;
      }
    }
  }
}

  //Start calculation with gpu
  sum = 0;
#pragma omp parallel \
  shared(dimGrid, dimBlock, nf, sum, dev_mem) private(f, B, B_d, C_d, C, gpuid, cpu_thread_id)
{

  checkCudaErrors(hipMalloc( (void **)&B_d, sizeof(int) * N*N));
  checkCudaErrors(hipMalloc( (void **)&C_d, sizeof(int) * N*N));


  checkCudaErrors(hipGetDevice(&gpuid));
  #pragma omp for reduction(+:sum)
  for (f=0; f<nf; f++)
  {
    fill_matrix(B, f+1, N, N);
    checkCudaErrors(hipMemcpy( B_d, B, sizeof(int) * N*N, hipMemcpyHostToDevice));
    //perform_operation(A, B, C, N, N);

    perform_operation_cuda<<<dimGrid, dimBlock>>>(dev_mem[gpuid].A_d, B_d, C_d, N, N);
    checkCudaErrors(hipMemcpy( C, C_d, sizeof(int) * N*N, hipMemcpyDeviceToHost));
    sum = do_sum(C, sum, N, N);
  }
  checkCudaErrors(hipFree(B_d));
  checkCudaErrors(hipFree(C_d));
}

  for (f = 0;f<num_gpus;f++)
  {
    checkCudaErrors(hipFree(dev_mem[f].A_d));
  }


  //check
  printf("SUM = %d\n", sum);
  printf("SUM - SUM_REF = %d\n", sum-sum_ref);
  checkCudaErrors(hipDeviceReset());

  return 0;
}